#include "hip/hip_runtime.h"
/*
 * @Author: Kaustav Vats 
 * @Roll-Number: 2016048 
 * @Date: 2019-01-22 23:03:14 
 */

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include <iostream>
#include "stb_image.h"
#include "stb_image_write.h"
#include <stdio.h>

#define maskCols 3
#define maskRows 3
#define imgchannels 1

#define TILE_SIZE 32

using namespace std;

void sequentialConvolution(const unsigned char*inputImage,const float * kernel ,unsigned char * outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels)
{
    int i, j, m, n, mm, nn;
    int kCenterX, kCenterY;                         // center index of kernel
    float sum;                                      // accumulation variable
    int rowIndex, colIndex;                         // indice di riga e di colonna

    const unsigned char * inputImageData = inputImage;
    kCenterX = kernelSizeX / 2;
    kCenterY = kernelSizeY / 2;

    // cout << kCenterX << " " << kCenterY << endl;

    for (int k=0; k<channels; k++) {                    //cycle on channels
        for (i = 0; i < dataSizeY; ++i)                //cycle on image rows
        {
            for (j = 0; j < dataSizeX; ++j)            //cycle on image columns
            {
                sum = 0;
                for (m = 0; m < kernelSizeY; ++m)      //cycle kernel rows
                {
                    mm = kernelSizeY - 1 - m;       // row index of flipped kernel

                    for (n = 0; n < kernelSizeX; ++n)  //cycle on kernel columns
                    {
                        nn = kernelSizeX - 1 - n;   // column index of flipped kernel

                        // indexes used for checking boundary
                        rowIndex = i + m - kCenterY;
                        colIndex = j + n - kCenterX;

                        // ignore pixels which are out of bound
                        if (rowIndex >= 0 && rowIndex < dataSizeY && colIndex >= 0 && colIndex < dataSizeX)
                            sum += inputImageData[(dataSizeX * rowIndex + colIndex)*channels + k] * kernel[kernelSizeX * mm + nn];
                    }
                }
                outputImageData[(dataSizeX * i + j)*channels + k] = sum;

            }
        }
    }
}
__global__ void convKernel(unsigned char * inputImage, const float * kernel, unsigned char* outputImageData, int kernelSizeX, int kernelSizeY, int dataSizeX, int dataSizeY, int channels){

    int m, n, mm, nn;
    int kCenterX, kCenterY;                         // center index of kernel
    float sum;                                      // accumulation variable
    int rowIndex, colIndex;                         // indice di riga e di colonna

    const unsigned char * inputImageData = inputImage;
    kCenterX = kernelSizeX / 2;
    kCenterY = kernelSizeY / 2;

    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    __shared__ unsigned char s_ImageData[TILE_SIZE][TILE_SIZE];

    // // __shared__ float s_Kernel[kernelSizeY][kernelSizeX];

    s_ImageData[i][j] = inputImageData[i * dataSizeX + j];

    __syncthreads();

    sum = 0;
    for (m = 0; m < kernelSizeY; ++m)      //cycle kernel rows
    {
        mm = kernelSizeY - 1 - m;       // row index of flipped kernel

        for (n = 0; n < kernelSizeX; ++n)  //cycle on kernel columns
        {
            nn = kernelSizeX - 1 - n;   // column index of flipped kernel

            // indexes used for checking boundary
            rowIndex = i + m - kCenterY;
            colIndex = j + n - kCenterX;

            // ignore pixels which are out of bound
            if (rowIndex >= 0 && rowIndex < dataSizeY && colIndex >= 0 && colIndex < dataSizeX)
                // sum += inputImageData[dataSizeX * rowIndex + colIndex] * kernel[kernelSizeX * mm + nn];
                sum += s_ImageData[rowIndex][colIndex] * kernel[kernelSizeX * mm + nn];
        }
    }
    outputImageData[dataSizeX * i + j] = sum;
}

int main(){
    int width, height, bpp;
    unsigned char *image, *seq_img;

    image = stbi_load( "image64.png", &width, &height, &bpp, imgchannels );
    seq_img = (unsigned char*)malloc(width*height*sizeof(unsigned char));

    cout << "Height x Width " << height << "x" << width << endl; 
 
    float hostMaskData[maskRows*maskCols];
    for(int i=0; i< maskCols*maskCols; i++){
        hostMaskData[i] = 1.0/(maskRows*maskCols);
    }
    // sequentialConvolution(image, hostMaskData, seq_img, maskRows, maskCols, width, height, imgchannels);
    // stbi_write_png("mynew_seq.png", width, height, imgchannels, seq_img, 0);

    // cuda Program

    const dim3 block_size(16, 16);
    const dim3 num_blocks(width/block_size.x, height/block_size.y);

    cout << "Block Size " << block_size.x << "x" << block_size.y << endl;
    cout << "Num Block " << num_blocks.x << "x" << num_blocks.y << endl;

    unsigned char *d_image = 0, *d_seqimg = 0;
    float *d_hostmaskdata = 0;

    hipMalloc((void**)&d_image, sizeof(unsigned char) * width * height);
    hipMalloc((void**)&d_seqimg, sizeof(unsigned char) * width * height);
    hipMalloc((void**)&d_hostmaskdata, sizeof(float) * maskCols * maskRows);

    hipMemcpy(d_image, &image[0], sizeof(char) * width * height, hipMemcpyHostToDevice);
    hipMemcpy(d_hostmaskdata, &hostMaskData[0], sizeof(float) * maskCols * maskRows, hipMemcpyHostToDevice);

    convKernel<<<num_blocks, block_size>>>(d_image, d_hostmaskdata, d_seqimg, maskRows, maskCols, width, height, imgchannels);

    hipMemcpy(seq_img, d_seqimg, sizeof(char) * width * height, hipMemcpyDeviceToHost);

    stbi_write_png("mynew_seq.png", width, height, imgchannels, seq_img, 0);    

    hipFree(d_image);
    hipFree(d_seqimg);
    hipFree(d_hostmaskdata);

    free(image);
    free(seq_img);

    return 0;
}