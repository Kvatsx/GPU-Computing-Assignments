#include "hip/hip_runtime.h"
/*
 * @Author: Kaustav Vats 
 * @Roll-Number: 2016048 
 */

 #include <iostream>
 #include <stdio.h>

using namespace std;

__global__ void matchPattern_GPU(unsigned int *text, unsigned int *words, int *matches, int nwords, int length) {

    // __shared__ int sm_matches[nwords];
    __shared__ unsigned int sm_text[blockDim.x+1][blockDim.y];
    
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    int total_threads = gridDim.x*gridDim.y*blockDim.x*blockDim.y;
    int index = row*gridDim.x + col;

    sm_text[col][row] = text[index];

    unsigned int word;
    while ( index < length ) {
        for (int offset=0; offset<4; offset++)
        {
            if (offset==0) {
                word = text[index];
                // cout << "wd:" << word << endl;
            }
            else {
                word = (text[l]>>(8*offset)) + (text[l+1]<<(32-8*offset)); 
            }

            for (int w=0; w<nwords; w++){
                // sm_matches[w] += (word==words[w]);
                matches[w] += (word==words[w]);
            } 
            // cout << "word : " << offset << word << endl;           
        }
        index += total_threads;
    }
}

int main() {
    
}
