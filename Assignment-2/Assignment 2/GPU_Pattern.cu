/*
 * @Author: Kaustav Vats 
 * @Roll-Number: 2016048 
 */

 
#include <hip/hip_runtime.h>
#include <iostream>
 #include <stdio.h>

using namespace std;

#define LINEWIDTH 20
// #define TOTAL 1025

__global__ void matchPattern_GPU(unsigned int *text, unsigned int *words, int *matches, int nwords, int length) {
    // printf("Kernel Called\n");
    // __shared__ int sm_matches[nwords];
    // __shared__ unsigned int sm_text[blockDim.x+1][blockDim.y];

    int row = blockIdx.y*blockDim.y + threadIdx.y;
    int col = blockIdx.x*blockDim.x + threadIdx.x;

    int total_threads = gridDim.x*gridDim.y*blockDim.x*blockDim.y;

    // __shared__ unsigned int sm_text[1025];

    int xDist = threadIdx.x + blockIdx.x*blockDim.x;
    int yDist = threadIdx.y*blockDim.x*gridDim.x + blockDim.y*blockIdx.y*blockDim.x*gridDim.x;
    int index = xDist + yDist;

    int i=0;
    unsigned int word;
    while ( (index + i*total_threads) < length ) {
        // sm_text[index] = text[index + i*total_threads];
        // if ( threadIdx.x == (blockDim.x-1) && threadIdx.y == (blockDim.y-1)) {
            // sm_text[total_threads] = text[index + i*total_threads+1];
        // }
        // __syncthreads();

        for (int offset=0; offset<4; offset++)
        {
            if (offset==0) {
                word = text[index];
                // word = sm_text[index];
            }
            else {
                word = (text[index]>>(8*offset)) + (text[index+1]<<(32-8*offset)); 
                // word = (sm_text[index]>>(8*offset)) + (sm_text[index+1]<<(32-8*offset)); 
            }

            for (int w=0; w<nwords; w++){
                if (word == words[w]) {
                    atomicAdd(&matches[w], 1);
                }
            }        
        }
        i++;
        // index += total_threads;
    }
}

int main() {

    int length, len, nwords=20, matches[nwords];
	char *ctext, keywords[nwords][LINEWIDTH], *line;
	line = (char*) malloc(sizeof(char)*LINEWIDTH);
	unsigned int  *text,  *words;
	memset(matches, 0, sizeof(matches));

	// char filename[1024];
	// memset(filename, '\0', sizeof(filename));

	// cin >> filename;
	// read in text and keywords for processing
	FILE *fp, *wfile;
	// wfile = fopen("./data/test.txt","r");
	wfile = fopen("./data/keywords.txt","r");
	if (!wfile)
	{	
		printf("keywords.txt: File not found.\n");	
		exit(0);
	}

	int k=0, cnt = nwords;
	size_t read, linelen = LINEWIDTH;
	while((read = getline(&line, &linelen, wfile)) != -1 && cnt--)
	{
		strncpy(keywords[k], line, sizeof(line));
		keywords[k][4] = '\0';
		k++;
	}
	fclose(wfile);

	// cout << "K: " << k << endl;
	fp = fopen("./data/small.txt","r");
	// fp = fopen("./data/small.txt","r");
	if (!fp) {	
		printf("Unable to open the file.\n");	
		exit(0);
	}

	length = 0;
	while (getc(fp) != EOF) length++;

	ctext = (char *) malloc(length+4);

	rewind(fp);

	for (int l=0; l<length; l++) ctext[l] = getc(fp);
	for (int l=length; l<length+4; l++) ctext[l] = ' ';

	fclose(fp);

	printf("Length : %d\n", length );
	// define number of words of text, and set pointers
	len  = length/4;
	text = (unsigned int *) ctext;

	// define words for matching
	words = (unsigned int *) malloc(nwords*sizeof(unsigned int));

	// cout << "Words: ";
	for (int w=0; w<nwords; w++)
	{
		words[w] = ((unsigned int) keywords[w][0])
             + ((unsigned int) keywords[w][1])*(1<<8)
             + ((unsigned int) keywords[w][2])*(1<<16)
             + ((unsigned int) keywords[w][3])*(1<<24);
		// cout << words[w] << "\t";
	}
	// cout << endl;

    // GPU Execution
    const dim3 block_size(32, 32);
    const dim3 num_blocks(1024, 1024);

    unsigned int * d_text, * d_words;
    int * d_matches;
    int * mat;
    mat = (int* )malloc(sizeof(int)*nwords);

    hipMalloc((void**)&d_text, sizeof(unsigned int) * len);
    hipMalloc((void**)&d_words, sizeof(unsigned int)*nwords);
    hipMalloc((void**)&d_matches, sizeof(int)*nwords);

    hipMemcpy(d_text, text, sizeof(unsigned int) * len, hipMemcpyHostToDevice);
    hipMemcpy(d_words, words, sizeof(unsigned int) * nwords, hipMemcpyHostToDevice);

    matchPattern_GPU<<<num_blocks, block_size>>>(d_text, d_words, d_matches, nwords, len);

    hipMemcpy(mat, d_matches, sizeof(int) * nwords, hipMemcpyDeviceToHost);


	// CPU execution
	// const clock_t begin_time = clock();
	// matchPattern_CPU(text, words, matches, nwords, len);
	// float runTime = (float)( clock() - begin_time ) /  CLOCKS_PER_SEC;
	// printf("Time for matching keywords: %fs\n\n", runTime);

	printf("Printing Matches:\n");
	printf("Word\t  |\tNumber of Matches\n===================================\n");
	for (int i = 0; i < nwords; ++i)
		printf("%s\t  |\t%d\n", keywords[i], mat[i]);

	free(ctext);
    free(words);
    hipFree(d_text);
    hipFree(d_words);
    hipFree(d_matches);

}
