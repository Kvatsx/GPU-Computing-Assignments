/*
 * @Author: Kaustav Vats 
 * @Roll-Number: 2016048 
 * @Ref:- CPU Code by TA
 */

 
#include <hip/hip_runtime.h>
#include <iostream>
 #include <stdlib.h>
 #include <stdio.h>
 #include <string.h>
 #include <math.h>
 #include <ctime>

using namespace std;

#define LINEWIDTH 20
#define TOTAL 32
#define MAX_WORDS 32

void CheckDiff(int * cpu, int * gpu, int nwords) {
    int Correct = 0, Wrong = 0;
    for (int i=0; i<nwords; i++) {
        if ( cpu[i] == gpu[i] ) {
            Correct++;
        }
        else {
            Wrong++;
        }
    }
    if (Wrong == 0) {
        printf("Correctly Matched: True\n\n");
    }
    else {
        printf("Correctly Matched: False\n\n");
    }
}

void matchPattern_CPU(unsigned int *text, unsigned int *words, int *matches, int nwords, int length)
{
	unsigned int word;

	for (int l=0; l<length; l++)
	{
		for (int offset=0; offset<4; offset++)
		{
			if (offset==0) {
				word = text[l];
			}
			else
				word = (text[l]>>(8*offset)) + (text[l+1]<<(32-8*offset)); 

			for (int w=0; w<nwords; w++){
				matches[w] += (word==words[w]);
			} 
		}
	}
}

__global__ void matchPattern_GPU(unsigned int *text, const unsigned int *words, int *matches, int nwords, int length) {

    int col = threadIdx.x;
    int index = col + blockIdx.x*blockDim.x;

    if (index >= length) {
        return;
    }

    __shared__ unsigned int sm_text[TOTAL+1];
    __shared__ unsigned int sm_words[MAX_WORDS];

    // int xDist = threadIdx.x + blockIdx.x*blockDim.x;
    // int yDist = threadIdx.y*blockDim.x*gridDim.x + blockDim.y*blockIdx.y*blockDim.x*gridDim.x;
    // int index = xDist + yDist;
    // __shared__ int len;

    // if (threadIdx.x == 0) {
    //     len = length;
    // }

    unsigned int word;

    if (col < 32) {
        sm_words[col] = words[col];
    }

    // if (index < length) {
        sm_text[col] = text[index];
        // if (col == TOTAL-1) {
        sm_text[col+1] = text[index+1];
        // }
    // }
    __syncthreads();

    for (int offset=0; offset<4; offset++)
    {
        if (offset==0) {
            // word = text[index];
            word = sm_text[col];
        }
        else {
            // word = (text[index]>>(8*offset)) + (text[index+1]<<(32-8*offset)); 
            word = (sm_text[col]>>(8*offset)) + (sm_text[col+1]<<(32-8*offset)); 
        }
        for (int w=0; w<32; w++){
            if (word == sm_words[w]) {
            // if (word == words[w]) {
                atomicAdd(&matches[w], 1);
            }
        }        
    }
}

int main() {

    const char * fileData[3] = { "./data/small.txt", "./data/medium.txt", "./data/large.txt" };
    int wl;
    for (wl=0; wl<3; wl++) {

        int length, len, nwords=32, matches[nwords];
        char *ctext, keywords[nwords][LINEWIDTH], *line;
        line = (char*) malloc(sizeof(char)*LINEWIDTH);
        unsigned int  *text,  *words;
        memset(matches, 0, sizeof(matches));

        // char filename[1024];
        // memset(filename, '\0', sizeof(filename));

        // cin >> filename;
        // read in text and keywords for processing
        FILE *fp, *wfile;
        // wfile = fopen("./data/test.txt","r");
        wfile = fopen("./data/keywords.txt","r");
        if (!wfile)
        {	
            printf("keywords.txt: File not found.\n");	
            exit(0);
        }

        int k=0, cnt = nwords;
        size_t read, linelen = LINEWIDTH;
        while((read = getline(&line, &linelen, wfile)) != -1 && cnt--)
        {
            strncpy(keywords[k], line, sizeof(line));
            keywords[k][4] = '\0';
            k++;
        }
        fclose(wfile);

        // cout << "K: " << k << endl;
        fp = fopen(fileData[wl],"r");
        // fp = fopen("./data/small.txt","r");
        if (!fp) {	
            printf("Unable to open the file.\n");	
            exit(0);
        }

        length = 0;
        while (getc(fp) != EOF) length++;

        ctext = (char *) malloc(length+4);

        rewind(fp);

        for (int l=0; l<length; l++) ctext[l] = getc(fp);
        for (int l=length; l<length+4; l++) ctext[l] = ' ';

        fclose(fp);

        printf("Length : %d\n", length );
        // define number of words of text, and set pointers
        len  = length/4;
        text = (unsigned int *) ctext;

        // define words for matching
        words = (unsigned int *) malloc(nwords*sizeof(unsigned int));

        // cout << "Words: ";
        for (int w=0; w<nwords; w++)
        {
            words[w] = ((unsigned int) keywords[w][0])
                + ((unsigned int) keywords[w][1])*(1<<8)
                + ((unsigned int) keywords[w][2])*(1<<16)
                + ((unsigned int) keywords[w][3])*(1<<24);
            // cout << words[w] << "\t";
        }
        // cout << endl;

        // GPU Execution
        const dim3 block_size(TOTAL, 1);
        // const dim3 num_blocks(512, 512);
        const dim3 num_blocks(ceil(len/TOTAL), 1);

        hipEvent_t start_kernel, stop_kernel, m_kernel_start, m_kernel_stop;
        hipEventCreate(&start_kernel);
        hipEventCreate(&stop_kernel);
        hipEventCreate(&m_kernel_start);
        hipEventCreate(&m_kernel_stop);


        unsigned int * d_text, * d_words;
        int * d_matches;
        int * mat;
        mat = (int* )malloc(sizeof(int)*nwords);

        hipEventRecord(m_kernel_start);
        hipMalloc((void**)&d_text, sizeof(unsigned int) * len);
        hipMalloc((void**)&d_words, sizeof(unsigned int)*nwords);
        hipMalloc((void**)&d_matches, sizeof(int)*nwords);

        hipMemcpy(d_text, text, sizeof(unsigned int) * len, hipMemcpyHostToDevice);
        hipMemcpy(d_words, words, sizeof(unsigned int) * nwords, hipMemcpyHostToDevice);

        hipEventRecord(start_kernel);
        matchPattern_GPU<<<num_blocks, block_size>>>(d_text, d_words, d_matches, nwords, len);
        hipEventRecord(stop_kernel);
        
        hipEventSynchronize(stop_kernel);

        hipMemcpy(mat, d_matches, sizeof(int) * nwords, hipMemcpyDeviceToHost);
        hipEventRecord(m_kernel_stop);
        hipEventSynchronize(m_kernel_stop);

        float k_time, k2_time ;
        hipEventElapsedTime(&k_time, start_kernel, stop_kernel);
        hipEventElapsedTime(&k2_time, m_kernel_start, m_kernel_stop);
        cout  << "[" << wl << "] GPU-Kernel Time: " << k_time  << "ms" << endl;
        cout  << "[" << wl << "] GPU-Kernel + Memory Time: " << k2_time  << "ms" << endl;

        // CPU execution
        const clock_t begin_time = clock();
        matchPattern_CPU(text, words, matches, nwords, len);
        float runTime = (float)( clock() - begin_time ) /  CLOCKS_PER_SEC;
        // printf("CPU Time: %fs\n\n", runTime);
        cout  << "[" << wl << "] CPU-Time: " << runTime << "s" << endl;

        cout << "[" << wl << "] Speedup: " << (runTime*1000)/k_time << endl;
        cout << "[" << wl << "] Speedup with memory Transfer: " << (runTime*1000)/k2_time << endl;
        
        CheckDiff(matches, mat, nwords);

        // printf("[%d] Printing Matches:\n", wl);
        // printf("Word\t  |\tNumber of Matches\n===================================\n");
        // for (int i = 0; i < nwords; ++i)
            // printf("%s\t  |\t%d\n", keywords[i], mat[i]);

        free(ctext);
        free(words);
        hipFree(d_text);
        hipFree(d_words);
        hipFree(d_matches);
    }

    return 0;
}
